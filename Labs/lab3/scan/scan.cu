#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256

/*
 * In the starter code, the reference solution scan implementation above assumes that 
 * the input array's length (N) is a power of 2. 
 * In the cudaScan function, we solve this problem by 
 * rounding the input array length to the next power of 2 when 
 * allocating the corresponding buffers on the GPU. 
 */
// helper function to round an integer up to the next power of 2
// (n == 3 return 4), (n == 5 return 8)
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

static inline std::pair<int, int> calGridBlock(int N, int two_d) {
    // 注意到threadPerBlock == 512 == 2^9，totalThread一定是2的倍数（因为我们保证N是2的倍数）
    // 所以除了totalThread <= threadPerBlock外，blockNum = totalThread / threadPerBlock一定是整数
    const int threadPerBlock = 512;
    int two_dplus1 = 2 * two_d;
    int totalThread = N / two_dplus1;
    int blockNum;
    if (totalThread <= threadPerBlock) return {1, totalThread};
    blockNum = totalThread / threadPerBlock;
    return {blockNum, threadPerBlock};
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
__global__ void Parallel_Upsweep(int two_d, int *output) {
    int two_dplus1 = 2 * two_d;
    int i = two_dplus1 * (blockIdx.x * blockDim.x + threadIdx.x);
    output[i + two_dplus1 - 1] += output[i + two_d - 1];
}

__global__ void Set_Zero(int N, int *output) {
    output[N - 1] = 0;
}

__global__ void Parallel_Downsweep(int two_d, int *output) {
    int two_dplus1 = 2 * two_d;
    int i = two_dplus1 * (blockIdx.x * blockDim.x + threadIdx.x);
    int t = output[i + two_d - 1];
    output[i + two_d - 1] = output[i + two_dplus1 - 1];
    output[i + two_dplus1 - 1] += t;
}

void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.
    /*
     * a naive implementation of scan might launch N CUDA threads for each iteration of 
     * the parallel loops in the pseudocode, and using conditional execution in the kernel 
     * to determine which threads actually need to do work. 
     * Such a solution will not be performant! (Consider the last outmost loop iteration of 
     * the upsweep phase, where only two threads would do work!). 
     * A full credit solution will only launch one CUDA thread for each iteration of 
     * the innermost parallel loops.
     */
     N = nextPow2(N);

     for (int two_d = 1; two_d <= N / 2; two_d *= 2) {
        auto [blockNum, threadPerBlock] = calGridBlock(N, two_d);
        Parallel_Upsweep<<<blockNum, threadPerBlock>>>(two_d, result);
        hipDeviceSynchronize();
    }

    Set_Zero<<<1, 1>>>(N, result);
    hipDeviceSynchronize();

    for (int two_d = N / 2; two_d >= 1; two_d /= 2) {
        auto [blockNum, threadPerBlock] = calGridBlock(N, two_d);
        Parallel_Downsweep<<<blockNum, threadPerBlock>>>(two_d, result);
        hipDeviceSynchronize();
    }
}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    /*
     * When calling your exclusive_scan implementation, 
     * remember that the contents of the start array are copied over to the output array. 
     * Also, the arrays passed to exclusive_scan are assumed to be in device memory. 
     */
    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}

__global__ void findRepeat_Flag(int *input, int N, int *output) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < N - 1) output[i] = (input[i] == input[i + 1]);
}

__global__ void findRepeat_Set(int *flags, int *idxs, int N, int *output) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N - 1 && flags[i] == 1) output[idxs[i]] = i;
}
// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.
    int res;
    int *device_flag, *device_idx;
    int rounded_length = nextPow2(length);
    hipMalloc((void **)&device_flag, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_idx, sizeof(int) * rounded_length);

    const int threadPerBlock = 512;
    int blockNum = (length + threadPerBlock - 1) / threadPerBlock;
    // GPU并行计算数组元素是否满足device_input[i] == device_input[i + 1]
    findRepeat_Flag<<<blockNum, threadPerBlock>>>(device_input, length, device_flag);
    hipDeviceSynchronize();

    hipMemcpy(device_idx, device_flag, sizeof(int) * length, hipMemcpyDeviceToDevice);
    // 对device_flag求扩展前缀和
    // 最大问题是我们如何将符合device_input[i] == device_input[i + 1]的i并行地放入device_input中？
    // 通过对device_flag求扩展前缀和得到的device_idx就可以知道放入device_input下标了
    // 即满足device_flag[i] == 1时的i，在device_idx[i]中的值即是device_input的下标
    // 即if (device_flag[i] == 1) device_input[device_idx[i]] = i;
    exclusive_scan(device_flag, length, device_idx);
    hipDeviceSynchronize();

    findRepeat_Set<<<blockNum, threadPerBlock>>>(device_flag, device_idx, length, device_output);

    hipMemcpy(&res, device_idx + length - 1, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_flag);
    hipFree(device_idx);
    return res; 
}


/*
 * Grading: We will test your code for correctness and performance on random input arrays.
 */
//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
